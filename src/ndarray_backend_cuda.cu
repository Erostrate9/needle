#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256
// #define TILE 4
#define TILE 16
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);
typedef ssize_t ptrdiff_t;

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  int32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<int32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides




__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  ssize_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN YOUR SOLUTION
  if (gid < size){
    size_t idx = 0, id = gid;
    for (size_t i = shape.size; i>0; --i){
        idx += (id % shape.data[i-1]) * strides.data[i-1];
        id /= shape.data[i-1];
    }
    out[gid] = a[offset + idx];
  }
  /// END YOUR SOLUTION
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
             std::vector<int32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}


__global__ void EwiseSetitemKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the EwiseSetitem operation.
   *
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN YOUR SOLUTION
  if (gid < size){
    size_t idx = 0, id = gid;
    for (size_t i = shape.size; i>0; --i){
        idx += (id % shape.data[i-1]) * strides.data[i-1];
        id /= shape.data[i-1];
    }
    out[offset + idx] = a[gid];
  }
  /// END YOUR SOLUTION
}

void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
                  std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  /// BEGIN YOUR SOLUTION
    CudaDims dim = CudaOneDim(out->size);
    EwiseSetitemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
  /// END YOUR SOLUTION
}


__global__ void ScalarSetitemKernel(const scalar_t val, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the ScalarSetitem operation.
   *
   * Args:
   *   size: number of elements to write in out array (note that this will not be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the
   *         product of items in shape, but convenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN YOUR SOLUTION
  if (gid < size){
    size_t idx = 0, id = gid;
    for (size_t i = shape.size; i>0; --i){
        idx += (id % shape.data[i-1]) * strides.data[i-1];
        id /= shape.data[i-1];
    }
    out[offset + idx] = val;
  }
  /// END YOUR SOLUTION
}

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<int32_t> shape,
                   std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  /// BEGIN YOUR SOLUTION
    CudaDims dim = CudaOneDim(out->size);
    ScalarSetitemKernel<<<dim.grid, dim.block>>>(val, out->ptr, size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */

/// BEGIN YOUR SOLUTION
// macros BEGIN
#define KERNEL_LEFT size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
  if (gid < size) out[gid]

#define INIT_DIM CudaDims dim = CudaOneDim(out->size)

#define EWISE_BINARY_KERNEL <<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size)

#define EWISE_UNARY_KERNEL <<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size)

#define SCALAR_BINARY_KERNEL <<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size)

// macros END

//  *   - EwiseMul, ScalarMul
__global__ void EwiseMulKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
    KERNEL_LEFT = a[gid] * b[gid];
}
void EwiseMul(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Multiply together two CUDA array
   */
  INIT_DIM;
  EwiseMulKernel EWISE_BINARY_KERNEL;
}

__global__ void ScalarMulKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
    KERNEL_LEFT = a[gid] * val;
}
void ScalarMul(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  INIT_DIM;
  ScalarMulKernel SCALAR_BINARY_KERNEL;
}

//  *   - EwiseDiv, ScalarDiv
__global__ void EwiseDivKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
    KERNEL_LEFT = a[gid] / b[gid];
}
void EwiseDiv(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  INIT_DIM;
  EwiseDivKernel EWISE_BINARY_KERNEL;
}

__global__ void ScalarDivKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
    KERNEL_LEFT = a[gid] / val;
}
void ScalarDiv(const CudaArray& a, scalar_t val, CudaArray* out) {
  INIT_DIM;
  ScalarDivKernel SCALAR_BINARY_KERNEL;
}
//  *   - ScalarPower
__global__ void ScalarPowerKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
    KERNEL_LEFT = powf(a[gid], val);
}
void ScalarPower(const CudaArray& a, scalar_t val, CudaArray* out) {
  INIT_DIM;
  ScalarPowerKernel SCALAR_BINARY_KERNEL;
}
//  *   - EwiseMaximum, ScalarMaximum
__global__ void EwiseMaximumKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
    KERNEL_LEFT = max(a[gid], b[gid]);
}
void EwiseMaximum(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  INIT_DIM;
  EwiseMaximumKernel EWISE_BINARY_KERNEL;
}

__global__ void ScalarMaximumKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
    KERNEL_LEFT = max(a[gid], val);
}
void ScalarMaximum(const CudaArray& a, scalar_t val, CudaArray* out) {
  INIT_DIM;
  ScalarMaximumKernel SCALAR_BINARY_KERNEL;
}
//  *   - EwiseEq, ScalarEq
__global__ void EwiseEqKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
    KERNEL_LEFT = (a[gid] == b[gid]);
}
void EwiseEq(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  INIT_DIM;
  EwiseEqKernel EWISE_BINARY_KERNEL;
}

__global__ void ScalarEqKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
    KERNEL_LEFT = (a[gid] == val);
}
void ScalarEq(const CudaArray& a, scalar_t val, CudaArray* out) {
  INIT_DIM;
  ScalarEqKernel SCALAR_BINARY_KERNEL;
}
//  *   - EwiseGe, ScalarGe
__global__ void EwiseGeKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
    KERNEL_LEFT = (a[gid] >= b[gid]);
}
void EwiseGe(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  INIT_DIM;
  EwiseGeKernel EWISE_BINARY_KERNEL;
}

__global__ void ScalarGeKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
    KERNEL_LEFT = (a[gid] >= val);
}
void ScalarGe(const CudaArray& a, scalar_t val, CudaArray* out) {
  INIT_DIM;
  ScalarGeKernel SCALAR_BINARY_KERNEL;
}
//  *   - EwiseLog
__global__ void EwiseLogKernel(const scalar_t* a, scalar_t* out, size_t size) {
    KERNEL_LEFT = logf(a[gid]);
}
void EwiseLog(const CudaArray& a, CudaArray* out) {
  INIT_DIM;
  EwiseLogKernel EWISE_UNARY_KERNEL;
}
//  *   - EwiseExp
__global__ void EwiseExpKernel(const scalar_t* a, scalar_t* out, size_t size) {
    KERNEL_LEFT = expf(a[gid]);
}
void EwiseExp(const CudaArray& a, CudaArray* out) {
  INIT_DIM;
  EwiseExpKernel EWISE_UNARY_KERNEL;
}
//  *   - EwiseTanh
__global__ void EwiseTanhKernel(const scalar_t* a, scalar_t* out, size_t size) {
    KERNEL_LEFT = tanhf(a[gid]);
}
void EwiseTanh(const CudaArray& a, CudaArray* out) {
  INIT_DIM;
  EwiseTanhKernel EWISE_UNARY_KERNEL;
}

//  *   - EwiseSin
__global__ void EwiseSinKernel(const scalar_t* a, scalar_t* out, size_t size) {
    KERNEL_LEFT = sinf(a[gid]);
}
void EwiseSin(const CudaArray& a, CudaArray* out) {
  INIT_DIM;
  EwiseSinKernel EWISE_UNARY_KERNEL;
}

//  *   - EwiseCos
__global__ void EwiseCosKernel(const scalar_t* a, scalar_t* out, size_t size) {
    KERNEL_LEFT = cosf(a[gid]);
}
void EwiseCos(const CudaArray& a, CudaArray* out) {
  INIT_DIM;
  EwiseCosKernel EWISE_UNARY_KERNEL;
}
/// END YOUR SOLUTION

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////


__global__ void MatmulKernel_naive(const scalar_t* a, const scalar_t* b, scalar_t* out, const size_t M, const size_t N, const size_t P) {
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid < M*P){
        size_t row = gid / P;
        size_t col = gid % P;
        out[gid] = 0.0f;
        for (size_t k=0; k<N; ++k){
            out[gid] += a[row*N + k] * b[k * P + col];
        }
    }
}

__global__ void MatmulKernel_tiled(const scalar_t* a, const scalar_t* b, scalar_t* out, const size_t M, const size_t N, const size_t P) {
    size_t bidx = blockIdx.x, bidy = blockIdx.y,
           tidx = threadIdx.x, tidy = threadIdx.y;
    int x_range = static_cast<int>(bidx + 1) * TILE - M,
        y_range = static_cast<int>(bidy + 1) * TILE - P;
    if (x_range > 0) {
        a -= x_range * N;
        out -= x_range * P;
    }
    if (y_range > 0) {
        b -= y_range;
        out -= y_range;
    }
    a += bidx * TILE * N;
    b += bidy * TILE;
    out += (bidx * TILE) * P + (bidy * TILE);
    __shared__ scalar_t smemA[TILE][TILE], smemB[TILE][TILE];
    scalar_t accumu = 0.0f;
    for (int i = 0; i < N; i += TILE) {
        smemA[tidx][tidy] = (tidy + i < N) ? a[(tidx)*N + (tidy + i)] : 0.0f;
        smemB[tidx][tidy] = (tidx + i < N) ? b[(tidx + i) * P + tidy] : 0.0f;
        __syncthreads();
        for (int j = 0; j < TILE; j++) {
            accumu += smemA[tidx][j] * smemB[j][tidy];
        }
        __syncthreads();
    }
    out[tidx * P + tidy] = accumu;
}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN YOUR SOLUTION
  Fill(out,0.0f);
  if(M < TILE || P < TILE || N < TILE){
  // Threads per block: BASE_THREAD_NUM = 256
  // Blocks in each dimension: ceil( (float) M*P / BASE_THREAD_NUM)
    CudaDims dim = CudaOneDim(M*P);
    MatmulKernel_naive<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  }else{
    dim3 block(TILE, TILE);
    dim3 grid((M - 1) / TILE + 1, (P - 1) / TILE + 1);
    MatmulKernel_tiled<<<grid, block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  }
  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////

__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t* out, const size_t reduce_size, const size_t size){
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid >= size) return;
    scalar_t max_val = a[gid * reduce_size];
    for (size_t i = gid * reduce_size + 1; i < (gid+1)*reduce_size; ++i ){
        max_val = fmaxf(max_val, a[i]);
    }
    out[gid] = max_val;
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  INIT_DIM;
  ReduceMaxKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, reduce_size, out->size);
  /// END YOUR SOLUTION
}


__global__ void ReduceSumKernel(const scalar_t* a, scalar_t* out, const size_t reduce_size, const size_t size){
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid >= size) return;
    scalar_t sum_val = a[gid * reduce_size];
    for (size_t i = gid * reduce_size + 1; i < (gid+1)*reduce_size; ++i ){
        sum_val += a[i];
    }
    out[gid] = sum_val;
}

void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
    INIT_DIM;
    ReduceSumKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, reduce_size, out->size);
  /// END YOUR SOLUTION
}

__global__ void StackKernel(scalar_t **arr, size_t size, size_t total_size,
                            scalar_t *out) {
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid  < total_size){
        size_t no = gid / size;
        size_t offset = gid % size;
        out[gid] = arr[no][offset];
    }
}

void Stack(const std::vector<CudaArray *> arr, size_t size, CudaArray* out){
//    size is the product of arr[0].shape
    CudaDims dim = CudaOneDim(out->size);
    size_t n = arr.size();
    scalar_t **host_ptr = (scalar_t **)std::malloc(n * sizeof(arr[0]->ptr));
    if (host_ptr == 0)
        throw std::bad_alloc();
    for (size_t i = 0; i < n; ++i ){
        host_ptr[i] = arr[i]->ptr;
    }
    scalar_t **arr_ptr = nullptr;
    hipError_t error = hipMalloc(&arr_ptr, n * sizeof(arr[0] -> ptr));
    if (error!= hipSuccess)
        throw std::runtime_error(hipGetErrorString(error));
    error = hipMemcpy(arr_ptr, host_ptr, n*sizeof(arr[0]-> ptr),
                        hipMemcpyHostToDevice);
    if (error!= hipSuccess)
        throw std::runtime_error(hipGetErrorString(error));

    StackKernel<<<dim.grid, dim.block>>>(arr_ptr, size, out->size, out->ptr);
}

__global__ void SplitKernel(const scalar_t *A, size_t size, size_t total_size,
                            scalar_t **out) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < total_size) {
    int no = gid / size;
    int offset = gid % size;
    out[no][offset] = A[gid];
  }
}

void Split(const CudaArray &A, uint32_t size, std::vector<CudaArray *> out) {
  CudaDims dim = CudaOneDim(A.size);
  size_t n = out.size();

  // copy array of pointers to device
  scalar_t **host_ptr = (scalar_t **)std::malloc(n * sizeof(out[0]->ptr));
  if (host_ptr == 0)
    throw std::bad_alloc();
  for (int i = 0; i < n; ++i) {
    host_ptr[i] = out[i]->ptr;
  }

  scalar_t **arr_ptr = nullptr;
  hipError_t error = hipMalloc(&arr_ptr, n * sizeof(out[0]->ptr));
  if (error != hipSuccess)
    throw std::runtime_error(hipGetErrorString(error));
  error = hipMemcpy(arr_ptr, host_ptr, n * sizeof(out[0]->ptr),
                     hipMemcpyHostToDevice);
  if (error != hipSuccess)
    throw std::runtime_error(hipGetErrorString(error));

  SplitKernel<<<dim.grid, dim.block>>>(A.ptr, size, A.size, arr_ptr);
}

}  // namespace cuda
}  // namespace needle


PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);
  m.def("ewise_sin", EwiseSin);
  m.def("ewise_cos", EwiseCos);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);

  m.def("stack", Stack);
  m.def("split", Split);
}